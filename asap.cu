#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <ctime>
#include <stdio.h>

using namespace std;

struct cuda_exception 
{
    explicit cuda_exception(const char *err) : error_info(err) {}
    explicit cuda_exception(const string &err) : error_info(err) {}
    string what() const throw() { return error_info; }

    private:
    string error_info;
};

void checkCudaError(const char *msg) 
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        string error_info(msg);
        error_info += " : ";
        error_info += hipGetErrorString(err);
        throw cuda_exception(error_info);
    }
}

template<typename T>
void random_graph_matrices(T *M, T *N, size_t height, size_t width, int p = 2) 
{
    for(size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            // Set diagonals to zero
            if (i == j) {
                M[i * width + j] = 0;
            } else {
                // Add random infinities, around half the graph will be "infinite"
                int inf = rand() % 2;
                if (inf) {
                    M[i* width + j] = 100000;
                    N[i * width + j] = 100000;
                } else {
                    // Generate random number between 1 and p+1 to represent the current edge
                    int random = rand() % p + 1;
                    M[i * width + j] = random;
                    N[i * width + j] = random;
                } 
            }
        }
    }
}

template<typename T>
void print_matrix(const T *M, size_t height, size_t width)
{
    if (height >= 32 || width >= 32) {
        cout << "a matrix of height " << height << ", of width " << width << endl;
        return;
    }

    for(size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            cout << M[i * width + j] << "   ";
        }
        cout << endl;
    }
    cout << endl;
}

void serial_fw(int *A, size_t n) {
    for (int k = 0; k < n; k++) {
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                const unsigned int kj = k *n + j;
                const unsigned int ij = i*n + j;
                const unsigned int ik = i*n + k;

                int t1 = A[ik] + A[kj];
                int t2 = A[ij];
                A[ij] = (t1 < t2) ? t1: t2;
            }
        }
    }
}

#define BLOCK_SIZE 16

__global__ void min_plus_kernel(int *C, size_t n, size_t k) 
{
    const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i < n) && (j < n) && (k < n)) {
        const unsigned int kj = k *n + j;
        const unsigned int ij = i*n + j;
        const unsigned int ik = i*n + k;

        int t1 = C[ik] + C[kj];
        int t2 = C[ij];
        C[ij] = (t1 < t2) ? t1: t2;
    }
}

void floyd_warshall_gpu(int *C, size_t n)
{
    size_t mem_size = n * n * sizeof(int);

    int *Cd;
    hipMalloc((void **)&Cd, mem_size);
    checkCudaError("allocating GPU memory for matrix");
    hipMemcpy(Cd, C, mem_size, hipMemcpyHostToDevice);

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid_dim((n + block_size.x - 1) / block_size.x,
        (n + block_size.y - 1) / block_size.y);
    for (int k = 0; k < n; k++) {
        min_plus_kernel<<<grid_dim, block_size>>>(Cd, n, k);
        hipDeviceSynchronize();
        checkCudaError("call the matrix multiplication kernel");
    }
    hipMemcpy(C, Cd, mem_size, hipMemcpyDeviceToHost);

    hipFree(Cd);
}

double time_fw_gpu(int *C, size_t n) {
    clock_t time1 = clock();

    floyd_warshall_gpu(C, n);

    clock_t time2 = clock();
    return (time2 - time1) / double(CLOCKS_PER_SEC);
}

double time_fw_serial(int *C, size_t n) {
    clock_t time1 = clock();

    serial_fw(C, n);

    clock_t time2 = clock();
    return (time2 - time1) / double(CLOCKS_PER_SEC);
}

int main(int argc, char *argv[])
{
    int *W, *serial_W;
    int n;
    if (argc == 2) {
        n = atoi(argv[1]);
    } else if (argc == 1) {
        cout << "Please enter a value for n: " << endl;
        cin >> n;
    } else {
        cout << "Usage: ./asap {n}" << endl;
        return;
    }
    W = new int[n * n];
    serial_W = new int[n*n]; 
 
    try {  
        random_graph_matrices(W, serial_W, n, n, 10);
        if (n < 32) {
            print_matrix(W, n, n);
            print_matrix(serial_W, n, n);
        }
        
        cout << "GPU: " << time_fw_gpu(W, n) << endl;
        cout << "serial: " << time_fw_serial(serial_W, n) << endl;
    } catch (cuda_exception &err) {
        cout << err.what() << endl;
        delete [] W;
        delete [] serial_W;
        return EXIT_FAILURE;
    } catch (...) {
        delete [] W;
        delete [] serial_W;
        cout << "unknown exeception" << endl;
        return EXIT_FAILURE;
    }

    if (n < 32) {
        print_matrix(W, n, n);
        print_matrix(serial_W, n, n);
    }

    delete [] W;
    delete [] serial_W;
    return 0;
}
