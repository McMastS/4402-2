#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <ctime>

using namespace std;

struct cuda_exception 
{
    explicit cuda_exception(const char *err) : error_info(err) {}
    explicit cuda_exception(const string &err) : error_info(err) {}
    string what() const throw() { return error_info; }

    private:
    string error_info;
};

void checkCudaError(const char *msg) 
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        string error_info(msg);
        error_info += " : ";
        error_info += hipGetErrorString(err);
        throw cuda_exception(error_info);
    }
}

template<typename T>
void random_matrices(T *M, T *N, size_t height, size_t width, int p = 2) 
{
    for(size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            int random = rand() % p;
            M[i * width + j] = random;
            N[i * width + j] = random; 
        }
    }
}

template<typename T>
void print_matrix(const T *M, size_t height, size_t width)
{
    if (height >= 32 || width >= 32) {
        cout << "a matrix of height " << height << ", of width " << width << endl;
        return;
    }

    for(size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            cout << M[i * width + j] << "   ";
        }
        cout << endl;
    }
    cout << endl;
}

void serial_min_plus(int *A, size_t n) {
    
    for (int k = 0; k < n; k++) {
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                t1 = A[i][k] + A[k][j]; 
                t2 = A[i][j];
                A[i][j] = ((t1 < t2) ? t1 : t2; 
            }
        }
    }
}

#define BLOCK_SIZE 4

__global__ void min_plus_kernel(int *C, size_t n, size_t k) 
{
    const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i >= n) || (j >= n) || (k >= n)) return;

    const unsigned int kj = k *n + j;
    const unsigned int ij = i*n + j;
    const unsigned int ik = i*n + k;

    int t1 = C[ik] + C[kj];
    int t2 = C[ij];
    C[ij] = (t1 < t2) ? t1: t2;
}

void min_plus_gpu(int *C, size_t n)
{
    size_t mem_size = n * n * sizeof(int);

    int *Cd;
    hipMalloc((void **)&Cd, mem_size);
    checkCudaError("allocating GPU memory for matrix");
    hipMemcpy(Cd, C, mem_size, hipMemcpyHostToDevice);
    for (int k = 0; k < n; k++) {
        min_plus_kernel<<<n/BLOCK_SIZE, BLOCK_SIZE>>>(Cd, n, k);
    }

    hipMemcpy(C, Cd, mem_size, hipMemcpyDeviceToHost);

    hipFree(Cd);
}

int main()
{
    int *W;
    int n;
    cout << "Please enter a value for n: " << endl;
    cin >> n;
    
    assert(n % BLOCK_SIZE == 0);

    try {
        W = new int[n * n];
        serial_W = new int[n * n];
        random_matrix(W, serial_W, n, n);

        min_plus_gpu(W, n);
        serial_min_plus(serial_W, n);
    } catch (cuda_exception &err) {
        cout << err.what() << endl;
        delete [] W;
        delete [] serial_W;
        return EXIT_FAILURE;
    } catch (...) {
        delete [] W;
        delete [] serial_W;
        cout << "unknown exeception" << endl;
        return EXIT_FAILURE;
    }

    print_matrix(W, n, n);
    print_matrix(serial_W, n, n);

    delete [] W;
    delete [] serial_W;
    return 0;
}
